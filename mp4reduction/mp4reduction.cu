#include "hip/hip_runtime.h"
// MP Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];


/* usage:
//@@ TODO
*/

# pragma warning (disable:4819)
#define TIMER_OK

#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "../include/wb.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

/* general define for cuda */
#define threadColIndex threadIdx.x
#define threadRowIndex threadIdx.y
#define blockColIndex blockIdx.x
#define blockRowIndex blockIdx.y

/* specific define for this project */
#define BLOCK_SIZE 256

void total(float * input, float * output, int len) {
    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index
}

int main(int argc, char ** argv) {
    int ii;
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    numOutputElements = numInputElements / (BLOCK_SIZE<<1);
    if (numInputElements % (BLOCK_SIZE<<1)) {
        numOutputElements++;
    }
    hostOutput = (float*)malloc(numOutputElements * sizeof(float));

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here

    wbTime_stop(Copy, "Copying output memory to the CPU");

    /********************************************************************
    * Reduce output vector on the host
    * NOTE: One could also perform the reduction of the output vector
    * recursively and support any size input. For simplicity, we do not
    * require that for this lab.
    ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] += hostOutput[ii];
    }

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, 1);

    free(hostInput);
    free(hostOutput);

    return 0;
}
