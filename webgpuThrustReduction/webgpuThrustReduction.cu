#include <wb.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

int main(int argc, char **argv) {
    wbArg_t args;
    float total;
    float *hostInput; // The input 1D list
    int numInputElements; // number of elements in the input list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput =
        (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);

    // Declare and allocate thrust device input and output vectors
    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Insert code here

    wbTime_stop(GPU, "Allocating GPU memory.");

    // Execute vector addition
    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ Insert Code here

    wbTime_stop(Compute, "Doing the computation on the GPU");

    wbSolution(args, &total, 1);

    free(hostInput);

    return 0;
}
