#include "hip/hip_runtime.h"
#define TIMER_OK

#include "../include/wb.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

/* usage:
.\vectorAdd.exe .\mp1\0\input1.raw .\mp1\0\input0.raw .\mp1\0\output.raw
*/


__global__
void vecAdd(float * in1, float * in2, float * out, int len)
{
    //@@ Insert code to implement vector addition here
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len)
        out[i] = in1[i] + in2[i];
}

int main(int argc, char ** argv)
{
    wbArg_t args;
    hipError_t returned;
    int inputLength;
    int inputByteSize;

    float * hostInput1;
    float * hostInput2;
    float * hostOutput;

    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "input length: ", inputLength);
    inputByteSize = inputLength * sizeof(float);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    returned = hipMalloc(&deviceInput1, inputByteSize);
    if (returned != hipSuccess) wbLog(ERROR, "hipMalloc(&deviceInput1");
    returned = hipMalloc(&deviceInput2, inputByteSize);
    if (returned != hipSuccess) wbLog(ERROR, "hipMalloc(&deviceInput2");
    returned = hipMalloc(&deviceOutput, inputByteSize);
    if (returned != hipSuccess) wbLog(ERROR, "hipMalloc(&deviceOutput");
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    returned = hipMemcpy(deviceInput1, hostInput1, inputByteSize, hipMemcpyHostToDevice);
    if (returned != hipSuccess) wbLog(ERROR, "hipMemcpy(deviceInput1");
    returned = hipMemcpy(deviceInput2, hostInput2, inputByteSize, hipMemcpyHostToDevice);
    if (returned != hipSuccess) wbLog(ERROR, "hipMemcpy(deviceInput2");
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(inputByteSize / 256 + 1, 1, 1);
    dim3 DimBlock(256, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd <<<DimGrid, DimBlock >>>(deviceInput1, deviceInput2, deviceOutput, inputByteSize);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    returned = hipMemcpy(hostOutput, deviceOutput, inputByteSize, hipMemcpyDeviceToHost);
    if (returned != hipSuccess) wbLog(ERROR, "hipMemcpy(hostOutput");
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
